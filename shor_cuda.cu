#include "hip/hip_runtime.h"
#include <thrust/complex.h>

//GTX1070
#define SM				1 //1 .. 15
#define CUDA_PER_SM		128
#define THREAD_PER_SM 	256 //32X - maximium 1024, 256 is optimal number

#define FOR() for(int i = blockIdx.x*blockDim.x + threadIdx.x; !(i>>n) && i < (1<<n); i += blockDim.x * gridDim.x) 

using cudouble = thrust::complex<double>;

__global__ void cuda_prepare_state(cudouble *data, int n, int period, cudouble amp) {
	FOR() {
		data[i] = (i %  (period) == 0 ? (amp) : 0.0);
	}
}

void gpu_prepare_state(int sm, cudouble *data, int n, int period) {
	const int total_period = ((1 << n) - 1) / period + 1;
	const cudouble amp = 1.0 / sqrt(total_period);

	cuda_prepare_state<<<sm, THREAD_PER_SM>>>(data, n, period, amp);
}

__global__ void cuda_hadamard(cudouble *data, int n, const cudouble sqrt_1_2, int mask_q) {
	FOR() {
		if (i & mask_q) continue;
		const int ii = i ^ mask_q;
		const cudouble a = sqrt_1_2 * (data[i] + data[ii]);
		const cudouble b = sqrt_1_2 * (data[i] - data[ii]);
		data[i] = a;
		data[ii] = b;
	}
}

void gpu_hadamard(int sm, cudouble *data, int n, int q) {
	static const cudouble sqrt_1_2 = sqrt(0.5);
	const int mask_q = 1 << q;
	cuda_hadamard<<<sm, THREAD_PER_SM>>>(data, n, sqrt_1_2, mask_q);
}

__global__ void cuda_controlled_rz(cudouble *data, int n, const cudouble omega, const int mask_q) {
	FOR() {
		if ((~i) & mask_q) continue;
		data[i] *= omega;
	}
}

void gpu_controlled_rz(int sm, cudouble *data, int n, const cudouble omega, const int mask_q) {
	cuda_controlled_rz<<<sm, THREAD_PER_SM>>>(data, n, omega, mask_q);
}

void gpu_init(cudouble **data, int n) {
	hipMalloc(data, sizeof(cudouble) * (1 << n));
}

void gpu_deinit(cudouble *data) {
	hipFree(data);
}

void gpu_memcpy(cudouble *dst, cudouble *src, int n) {
	hipMemcpy(dst, src, sizeof(cudouble) * (1 << n), hipMemcpyDeviceToHost);
}
